#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hostFE.h"
// for cuda
#include <hip/hip_runtime.h>

#define BlockSize 8

__constant__ float flt[1024];

__global__ void conv(float *in_img, float *out_img,
                     int img_w, int img_h, int flt_s, int pitch)
{
    int x = threadIdx.x + blockIdx.x * BlockSize;
    int y = threadIdx.y + blockIdx.y * BlockSize;
    if (x >= img_w || y >= img_h)
        return;

    int shift = flt_s / 2;
    float sum = 0;
    for (int i = 0; i < flt_s; i++)
    {
        for (int j = 0; j < flt_s; j++)
        {
            int nx = x - shift + j;
            int ny = y - shift + i;
            if ((nx < 0) || (nx >= img_w) || (ny < 0) || (ny >= img_h))
            {
                continue;
            }
            sum += flt[i * flt_s + j] * in_img[ny * pitch + nx];
        }
    }
    out_img[x + y * img_w] = sum;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(int filterWidth, float *filter, int imageHeight, int imageWidth,
            float *inputImage, float *outputImage)
{
    // allocate host mem
    float *in_img;
    size_t pitch;
    hipMallocPitch((void **)&in_img, &pitch, imageWidth * sizeof(float), imageHeight);

    hipHostRegister((void *)outputImage, imageWidth * imageHeight * sizeof(float),
                     hipHostRegisterMapped);

    float *out_img;
    hipHostGetDevicePointer(&out_img, outputImage, 0);
    // copy mem to device
    hipMemcpy2D(in_img, pitch, inputImage, imageWidth * sizeof(float),
                 imageWidth * sizeof(float), imageHeight, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(flt), filter, filterWidth * filterWidth * sizeof(float), 0,
                       hipMemcpyHostToDevice);

    dim3 dimBlock(BlockSize, BlockSize);
    dim3 dimGrid((imageWidth / BlockSize) + (imageWidth % BlockSize == 0 ? 0 : 1),
                 (imageHeight / BlockSize) + (imageHeight % BlockSize == 0 ? 0 : 1));

    conv<<<dimGrid, dimBlock>>>(in_img, out_img, imageWidth, imageHeight, filterWidth, pitch / sizeof(float));

    hipDeviceSynchronize();
    hipHostUnregister((void *)outputImage);
    hipFree(in_img);
    return;
}