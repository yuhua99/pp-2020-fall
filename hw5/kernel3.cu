#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BlockSize 16
#define GroupSize 4

__global__ void mandelKernel(int *cudaMem, float lowerX, float lowerY,
                             float stepX, float stepY, int maxIteration,
                             int widthX, int widthY, int pitch)
{
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int xid = (threadIdx.x + blockIdx.x * BlockSize) * GroupSize;
    int yid = threadIdx.y + blockIdx.y * BlockSize;
    if (yid >= widthY)
        return;

    float y = lowerY + yid * stepY;
    for (int i = 0; i < GroupSize; i++)
    {
        if (xid >= widthX)
            return;

        float x = lowerX + xid * stepX;
        float z_re = x, z_im = y;

        int j;
        for (j = 0; j < maxIteration; j++)
        {
            if (z_re * z_re + z_im * z_im > 4.f)
                break;
            float new_re = z_re * z_re - z_im * z_im;
            float new_im = 2.f * z_re * z_im;
            z_re = x + new_re;
            z_im = y + new_im;
        }
        cudaMem[xid + yid * pitch] = j;
        xid++;
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *Mem;
    hipHostAlloc((void **)&Mem, resX * resY * sizeof(int), hipHostMallocDefault);

    int *cudaMem;
    size_t pitch;
    hipMallocPitch((void **)&cudaMem, &pitch, resX * sizeof(int), resY);

    dim3 dimBlock(BlockSize, BlockSize);
    // group them
    dim3 dimGrid((resX / BlockSize * GroupSize) +
                     (resX % (BlockSize * GroupSize) == 0 ? 0 : 1),
                 (resY / BlockSize) + (resY % BlockSize == 0 ? 0 : 1));
    //  run on GPU
    mandelKernel<<<dimGrid, dimBlock>>>(cudaMem, lowerX, lowerY, stepX, stepY,
                                        maxIterations, resX, resY, pitch / sizeof(int));
    //  wait for work done
    hipDeviceSynchronize();
    hipMemcpy2D((void *)Mem, resX * sizeof(int), (void *)cudaMem, pitch,
                 resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy((void *)img, (void *)Mem, resX * resY * sizeof(int));
    // free mem
    hipFree(cudaMem);
    hipHostFree(Mem);
}