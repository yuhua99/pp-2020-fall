#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BlockSize 8

__global__ void mandelKernel(int *cudaMem, float lowerX, float lowerY,
                             float stepX, float stepY, int maxIteration,
                             int widthX, int widthY, int pitch)
{
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    // get pixel id
    int xid = threadIdx.x + blockIdx.x * BlockSize;
    int yid = threadIdx.y + blockIdx.y * BlockSize;
    if (xid >= widthX || yid >= widthY)
        return;

    float x = lowerX + xid * stepX;
    float y = lowerY + yid * stepY;
    float z_re = x, z_im = y;

    int i;
    for (i = 0; i < maxIteration; i++)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = x + new_re;
        z_im = y + new_im;
    }
    cudaMem[xid + yid * pitch] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    // allocate host mem
    int *Mem = (int *)malloc(resX * resY * sizeof(int));

    int *cudaMem;
    size_t pitch;
    hipMallocPitch((void **)&cudaMem, &pitch, resX * sizeof(int), resY);

    dim3 dimBlock(BlockSize, BlockSize);
    dim3 dimGrid((resX / BlockSize) + (resX % BlockSize == 0 ? 0 : 1),
                 (resY / BlockSize) + (resY % BlockSize == 0 ? 0 : 1));
    //  run on GPU
    mandelKernel<<<dimGrid, dimBlock>>>(cudaMem, lowerX, lowerY, stepX, stepY,
                                        maxIterations, resX, resY, pitch / sizeof(int));
    //  wait for work done
    hipDeviceSynchronize();
    hipMemcpy2D((void *)Mem, resX * sizeof(int), (void *)cudaMem, pitch,
                 resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy((void *)img, (void *)Mem, resX * resY * sizeof(int));
    // free mem
    hipFree(cudaMem);
    hipHostFree(Mem);
}