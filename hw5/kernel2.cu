#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BlockSize 16

__global__ void mandelKernel(int *cudaMem, float lowerX, float lowerY,
                             float stepX, float stepY, int maxIteration,
                             int widthX, int widthY, int pitch)
{
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int xid = threadIdx.x + blockIdx.x * BlockSize;
    int yid = threadIdx.y + blockIdx.y * BlockSize;
    if (xid >= widthX || yid >= widthY)
        return;

    float x = lowerX + xid * stepX;
    float y = lowerY + Yid * stepY;
    float z_re = x, z_im = y;

    int i;
    for (i = 0; i < maxIteration; i++)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = x + new_re;
        z_im = y + new_im;
    }
    cudaMem[xid + yid * pitch] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *Mem;
    hipHostAlloc((void **)&Mem, resX * resY * sizeof(int), hipHostMallocDefault);

    int *cudaMem;
    size_t pitch;
    hipMallocPitch((void **)&cudaMem, &pitch, rexX * sizeof(int), resY);
    fflush(stdout);

    dim3 dimBLock(BlockSize, BlockSize);
    dim3 dimGrid((resX / BLockSize) + (resX % BlockSize == 0 ? 0 : 1),
                 (resY / BLockSize) + (resY % BlockSize == 0 ? 0 : 1));
    //  run on GPU
    mandelKernel<<<dimGrid, dimBlock>>>(cudaMem, lowerX, lowerY, stepX, stepY,
                                        maxIterations, resX, resY, pitch / sizeof(int));
    //  wait for work done
    hipDeviceSynchronize();
    hipMemcpy2D((void *)Mem, resX * sizeof(int), (void *)cudaMem, pitch,
                 resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy((void *)img, (void *)Mem, resX * resY * sizeof(int));
    // free mem
    hipFree(cudaMem);
    hipHostFree(Mem);
}