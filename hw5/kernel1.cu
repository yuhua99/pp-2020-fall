#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BlockSize 16

__global__ void mandelKernel(int *cudaMem, float lowerX, float lowerY,
                             float stepX, float stepY, int maxIteration, int widthX, int widthY)
{
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int xid = threadIdx.x + blockIdx.x * BlockSize;
    int yid = threadIdx.y + blockIdx.y * BlockSize;
    if (xid >= widthX || yid >= widthY)
        return;

    float x = lowerX + xid * stepX;
    float y = lowerY + Yid * stepY;
    float z_re = x, z_im = y;

    int i;
    for (i = 0; i < maxIteration; i++)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = x + new_re;
        z_im = y + new_im;
    }
    cudaMem[xid + yid * widthX] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *Mem = (int *)malloc(resX * resY * sizeof(int));
    int *cudaMem;
    hipMalloc((void **)&cudaMem, resX * resY * sizeof(int));
    dim3 dimBLock(BlockSize, BlockSize);
    dim3 dimGrid((resX / BLockSize) + (resX % BlockSize == 0 ? 0 : 1),
                 (resY / BLockSize) + (resY % BlockSize == 0 ? 0 : 1));
    //  run on GPU
    mandelKernel<<<dimGrid, dimBlock>>>(cudaMem, lowerX, lowerY, stepX, stepY,
                                        maxIterations, resX, resY);
    //  wait for work done
    hipDeviceSynchronize();
    hipMemcpy((void *)Mem, (void *)cudaMem, resX * resY * sizeof(int),
               hipMemcpyDeviceToHost);
    memcpy((void *)img, (void *)Mem, resX * resY * sizeof(int));
    // free mem
    hipFree(cudaMem);
    free(Mem);
}